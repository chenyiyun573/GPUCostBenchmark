#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <nvml.h>
#include <stdio.h>
#include <pthread.h>
#include <vector>
#include <chrono>
#include <unistd.h>

#define ARRAY_SIZE      268435456   // 256M
#define BLOCK_SIZE      1024
#define UNROLL_DEPTH    128

#define SAMPLING_FREQUENCY 10 // 10Hz

static __device__ char ld_arr[ARRAY_SIZE];

static __device__ float sfma_out;
static __device__ double dfma_out;

struct PowerSamplingData {
    bool continueSampling;
    std::vector<unsigned int> powerData;
    nvmlDevice_t device;
    std::chrono::milliseconds samplingInterval;
};

void* powerSamplingThreadFunc(void* arg)
{
    PowerSamplingData* data = (PowerSamplingData*)arg;
    while (data->continueSampling) {
        unsigned int power;
        nvmlDeviceGetPowerUsage(data->device, &power);
        data->powerData.push_back(power);
        usleep(data->samplingInterval.count());
    }
    return nullptr;
}

__global__ void gmem_fp_hammer_kernel()
{
    int nd = ARRAY_SIZE >> 3;
    volatile double *ptr = (volatile double *)ld_arr;
    int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    int sz = BLOCK_SIZE * gridDim.x;
    int usz = sz * UNROLL_DEPTH;
    double x = 0;

    float sa = -1e5f;
    float sb = -1e5f;
    double da = -1e7;
    double db = -1e7;

    nd -= (nd % usz);
    __syncthreads();
    for (int it = 0; it < 1024; ++it) {
        for (int i = idx; i < nd; i += usz) {
            #pragma unroll
            for (int j = 0; j < UNROLL_DEPTH; ++j) {
                x += ptr[i+j*sz];
            }
            #pragma unroll
            for (int j = 0; j < 670; ++j) {
                asm ("fma.rn.f64    %0, %0, 0.9999, 0.01;" : "+d"(da));
                asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sa));
                asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sb));
                asm ("fma.rn.f64    %0, %0, 0.9999, 0.01;" : "+d"(db));
                asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sa));
                asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sb));
            }
        }
    }
    if (sa < 0) {
        // This is for avoiding compiler optimization.
        // Program never reach here.
        sfma_out = sa + sb;
        dfma_out = da + db;
        ptr[idx] = x;
    }
}

extern "C" {

hipError_t gmem_fp_hammer(hipStream_t s, int nblks)
{
    dim3 grid = dim3(nblks, 1, 1);
    dim3 block = dim3(BLOCK_SIZE, 1, 1);
    return hipLaunchKernel(reinterpret_cast<const void*>((void *)gmem_fp_hammer_kernel), grid, block, 0, 0, s);
}

int main() {
    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    PowerSamplingData powerSamplingData;
    powerSamplingData.continueSampling = true;
    powerSamplingData.device = device;
    powerSamplingData.samplingInterval = std::chrono::milliseconds(1000 / SAMPLING_FREQUENCY);

    // Create a new thread for power sampling
    pthread_t powerSamplingThread;
    pthread_create(&powerSamplingThread, NULL, powerSamplingThreadFunc, &powerSamplingData);

    // Keep sampling power for an additional 8 seconds before kernel execution
    sleep(8);

    // Call kernel
    hipStream_t stream;
    hipStreamCreate(&stream);
    gmem_fp_hammer(stream, 1024);
    hipDeviceSynchronize();

    // Keep sampling power for an additional 8 seconds after kernel execution
    sleep(16);

    // Stop power sampling
    powerSamplingData.continueSampling = false;
    pthread_join(powerSamplingThread, NULL);

    // Output to file
    FILE* file = fopen("powerData.csv", "w");
    fprintf(file, "Time(ms),Power(mW)\n");
    for (int i = 0; i < powerSamplingData.powerData.size(); i++) {
        fprintf(file, "%d,%d\n", i * (1000 / SAMPLING_FREQUENCY), powerSamplingData.powerData[i]);
    }
    fclose(file);

    nvmlShutdown();
    return 0;
}

} // extern "C"
