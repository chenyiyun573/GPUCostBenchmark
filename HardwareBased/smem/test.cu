#include <hip/hip_runtime.h>
#include <nvml.h>
#include <stdio.h>
#include <pthread.h>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <fstream>

#define CACHED_ARRAY_SIZE  49152    // 48KB
#define BLOCK_SIZE     640
#define SAMPLING_FREQUENCY 10 // 10Hz

struct PowerSamplingData {
    bool continueSampling;
    std::vector<unsigned int> powerData;
    nvmlDevice_t device;
    std::chrono::milliseconds samplingInterval;
};

void* powerSamplingThreadFunc(void* arg)
{
    PowerSamplingData* data = (PowerSamplingData*)arg;
    while (data->continueSampling) {
        unsigned int power;
        nvmlDeviceGetPowerUsage(data->device, &power);
        data->powerData.push_back(power);
        usleep(data->samplingInterval.count() * 1000);  // convert to microseconds
    }
    return nullptr;
}

static __global__ void smem_ld_hammer_kernel()
{
    __shared__ char arr[CACHED_ARRAY_SIZE];
    constexpr int ntmo = BLOCK_SIZE - 1;
    constexpr int nd = CACHED_ARRAY_SIZE / 8;
    double x = 0;
    int tid = threadIdx.x;
    for (int it = 0; it < 12000000; ++it) {
        double *ptr = (double *)arr;
        for (int i = 0; i < nd; i += BLOCK_SIZE) {
            #pragma unroll
            for (int j = 0; j < BLOCK_SIZE; j += 32) {
                int offset = (tid + j) & ntmo;
                x += ptr[offset];
            }
            ptr += 32;
        }
    }
    // For avoiding compiler optimization.
    ((double *)arr)[tid] = x;
}

extern "C" {

hipError_t smem_ld_hammer(hipStream_t s, int nblks)
{
    dim3 grid(nblks, 1, 1);
    dim3 block(BLOCK_SIZE, 1, 1);
    smem_ld_hammer_kernel<<<grid, block, 0, s>>>();
    return hipGetLastError();
}

} // extern "C"

int main() {
    // Initialize NVML
    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    // Initialize power sampling data
    PowerSamplingData powerSamplingData;
    powerSamplingData.continueSampling = true;
    powerSamplingData.device = device;
    powerSamplingData.samplingInterval = std::chrono::milliseconds(1000 / SAMPLING_FREQUENCY);

    // Create a new thread for power sampling
    pthread_t powerSamplingThread;
    pthread_create(&powerSamplingThread, NULL, powerSamplingThreadFunc, &powerSamplingData);

    // Keep sampling power for an additional 8 seconds before kernel execution
    sleep(8);

    // Call kernel
    hipStream_t stream;
    hipStreamCreate(&stream);
    smem_ld_hammer(stream, 1024);
    hipDeviceSynchronize();

    // Keep sampling power for an additional 8 seconds after kernel execution
    sleep(16);

    // Stop power sampling
    powerSamplingData.continueSampling = false;
    pthread_join(powerSamplingThread, NULL);

    // Output to file
    std::ofstream file("powerData.csv");
    file << "Time(ms),Power(mW)\n";
    for (int i = 0; i < powerSamplingData.powerData.size(); i++) {
        file << i * (1000 / SAMPLING_FREQUENCY) << "," << powerSamplingData.powerData[i] << "\n";
    }
    file.close();

    nvmlShutdown();
    return 0;
}
