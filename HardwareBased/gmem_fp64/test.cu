#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <nvml.h>
#include <stdio.h>
#include <pthread.h>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <fstream>

#define ARRAY_SIZE 268435456   // 256M
#define BLOCK_SIZE 1024
#define UNROLL_DEPTH 128
#define SAMPLING_FREQUENCY 50 // Hz

static __device__ char ld_arr[ARRAY_SIZE];
static __device__ float sfma_out;
static __device__ double dfma_out;

struct PowerSamplingData {
    bool continueSampling;
    std::vector<unsigned int> powerData;
    nvmlDevice_t device;
    std::chrono::milliseconds samplingInterval;
};

void* powerSamplingThreadFunc(void* arg)
{
    PowerSamplingData* data = (PowerSamplingData*)arg;
    while (data->continueSampling) {
        unsigned int power;
        nvmlDeviceGetPowerUsage(data->device, &power);
        data->powerData.push_back(power);
        usleep(data->samplingInterval.count() * 1000);  // convert to microseconds
    }
    return nullptr;
}

extern "C"
{
    __global__ void gmem_fp_hammer_kernel()
    {
        int nd = ARRAY_SIZE >> 3;
        volatile double *ptr = (volatile double *)ld_arr;
        int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
        int sz = BLOCK_SIZE * gridDim.x;
        int usz = sz * UNROLL_DEPTH;
        double x = 0;

        float sa = -1e5f;
        float sb = -1e5f;
        double da = -1e7;
        double db = -1e7;

        nd -= (nd % usz);
        __syncthreads();
        for (int it = 0; it < 1024; ++it) {
            for (int i = idx; i < nd; i += usz) {
                #pragma unroll
                for (int j = 0; j < UNROLL_DEPTH; ++j) {
                    x += ptr[i+j*sz];
                }
                #pragma unroll
                for (int j = 0; j < 670; ++j) {
                    asm ("fma.rn.f64    %0, %0, 0.9999, 0.01;" : "+d"(da));
                    asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sa));
                    asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sb));
                    asm ("fma.rn.f64    %0, %0, 0.9999, 0.01;" : "+d"(db));
                    asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sa));
                    asm ("fma.rn.f32    %0, %0, 0.9999, 0.01;" : "+f"(sb));
                }
            }
        }
        if (sa < 0) {
            // This is for avoiding compiler optimization.
            // Program never reach here.
            sfma_out = sa + sb;
            dfma_out = da + db;
            ptr[idx] = x;
        }
    }

    hipError_t gmem_fp_hammer(hipStream_t s, int nblks)
    {
        dim3 grid = dim3(nblks, 1, 1);
        dim3 block = dim3(BLOCK_SIZE, 1, 1);
        return hipLaunchKernel(reinterpret_cast<const void*>((void *)gmem_fp_hammer_kernel), grid, block, 0, 0, s);
    }
}


int main()
{
    // Set the number of blocks you want to launch
    int numBlocks = 1;

    // Initialize NVML
    nvmlReturn_t nvmlResult;
    nvmlResult = nvmlInit();
    if (nvmlResult != NVML_SUCCESS)
    {
        fprintf(stderr, "Failed to initialize NVML: %s\n", nvmlErrorString(nvmlResult));
        return 1;
    }

    // Query GPU device information
    nvmlDevice_t device;
    nvmlResult = nvmlDeviceGetHandleByIndex(0, &device); // Assuming one GPU is available
    if (nvmlResult != NVML_SUCCESS)
    {
        fprintf(stderr, "Failed to get GPU device handle: %s\n", nvmlErrorString(nvmlResult));
        nvmlShutdown();
        return 1;
    }

    // Allocate memory on the device for the workload
    hipError_t cudaStatus;
    hipStream_t stream;
    cudaStatus = hipStreamCreate(&stream);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipStreamCreate failed: %s\n", hipGetErrorString(cudaStatus));
        nvmlShutdown();
        return 1;
    }

    // Initialize power sampling data
    PowerSamplingData powerSamplingData;
    powerSamplingData.continueSampling = true;
    powerSamplingData.device = device;
    powerSamplingData.samplingInterval = std::chrono::milliseconds(1000 / SAMPLING_FREQUENCY);

    // Create a new thread for power sampling
    pthread_t powerSamplingThread;
    pthread_create(&powerSamplingThread, NULL, powerSamplingThreadFunc, &powerSamplingData);

    // Keep sampling power for an additional 10 seconds before kernel execution
    sleep(8);

    // Launch the workload kernel
    cudaStatus = gmem_fp_hammer(stream, numBlocks);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "fp32_hammer failed: %s\n", hipGetErrorString(cudaStatus));
        hipStreamDestroy(stream);
        nvmlShutdown();
        return 1;
    }

    // Synchronize the CUDA stream to ensure the kernel execution is completed
    cudaStatus = hipStreamSynchronize(stream);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipStreamSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
        hipStreamDestroy(stream);
        nvmlShutdown();
        return 1;
    }

    // Keep sampling power for an additional 10 seconds after kernel execution
    sleep(8);

    // Stop the power sampling thread
    powerSamplingData.continueSampling = false;
    pthread_join(powerSamplingThread, NULL);

    // Open CSV file
    std::ofstream outFile("power_data.csv");
    outFile << "Time(ms),Power(mW)\n";

    // Output the collected power data to the CSV file
    for (size_t i = 0; i < powerSamplingData.powerData.size(); ++i)
    {
        outFile << i * powerSamplingData.samplingInterval.count() << ","
                << powerSamplingData.powerData[i] << "\n";
    }

    // Close the CSV file
    outFile.close();

    // Free the allocated memory, destroy the CUDA stream, and shut down NVML
    hipStreamDestroy(stream);
    nvmlShutdown();

    return 0;
}

