// main.cu
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <stdio.h>
#include <pthread.h>
#include <vector>
#include <chrono>
#include <unistd.h>
#include <fstream>

#define SAMPLING_FREQUENCY 10 // 10Hz
#define FLOPS_PER_THREAD 33554432LL // Calculated from kernel

struct PowerSamplingData {
    bool continueSampling;
    std::vector<unsigned int> powerData;
    nvmlDevice_t device;
    std::chrono::milliseconds samplingInterval;
};

void* powerSamplingThreadFunc(void* arg)
{
    PowerSamplingData* data = (PowerSamplingData*)arg;
    while (data->continueSampling) {
        unsigned int power;
        nvmlDeviceGetPowerUsage(data->device, &power);
        data->powerData.push_back(power);
        usleep(data->samplingInterval.count() * 1000);  // convert to microseconds
    }
    return nullptr;
}

extern "C" hipError_t fp_hammer(hipStream_t s, int nblks);

int main() {
    // Initialize NVML
    nvmlInit();
    nvmlDevice_t device;
    nvmlDeviceGetHandleByIndex(0, &device);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // Initialize power sampling data
    PowerSamplingData powerSamplingData;
    powerSamplingData.device = device;
    powerSamplingData.samplingInterval = std::chrono::milliseconds(1000 / SAMPLING_FREQUENCY);

    // Create a new thread for power sampling
    pthread_t powerSamplingThread;
    pthread_create(&powerSamplingThread, NULL, powerSamplingThreadFunc, &powerSamplingData);

    // Loop over number of blocks
    for(int nblks = 2; nblks <= 1024*64; nblks *= 2) {
        // Reset power data and continue sampling flag for each iteration
        powerSamplingData.powerData.clear();
        powerSamplingData.continueSampling = true;

        // Keep sampling power for an additional 2 seconds before kernel execution
        sleep(2);

        // Start timing
        auto start = std::chrono::high_resolution_clock::now();

        // Call kernel
        fp_hammer(stream, nblks);
        hipDeviceSynchronize();

        // Stop timing
        auto stop = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

        // Keep sampling power for an additional 2 seconds after kernel execution
        sleep(2);

        // Stop power sampling
        powerSamplingData.continueSampling = false;

        // Calculate average power
        // Calculate average power
        unsigned int sumPower = 0;
        int powerDataSize = powerSamplingData.powerData.size();
        int numSamplesToSkip = 2 * SAMPLING_FREQUENCY; // Number of samples during 2 seconds
        for (int i = numSamplesToSkip; i < powerDataSize - numSamplesToSkip; i++)
            sumPower += powerSamplingData.powerData[i];
        double avgPower = static_cast<double>(sumPower) / (powerDataSize - 2 * numSamplesToSkip) / 1000; // convert to watts

        double tflops = (static_cast<double>(FLOPS_PER_THREAD * nblks * 640)) / (duration.count() * 1e6); // TFLOPs



        printf("nblks: %i, Execution Time: %ld us, Performance: %lf TFLOPS, Average Power: %lf W, Power Efficiency: %lf TFLOPS/W\n", nblks,
                duration.count(), tflops, avgPower, tflops / avgPower);
        
        
    }

    // Join the power sampling thread after all kernel launches
    pthread_join(powerSamplingThread, NULL);

    nvmlShutdown();
    return 0;
}
